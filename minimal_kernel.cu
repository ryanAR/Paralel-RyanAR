#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

void incrementArrayOnHost(float *a, int N)
{
    int i;
    for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<N) a[idx] = a[idx]+1.f;
}

__global__ void kernel_a (float *a)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    a[idx] = 7;
}

__global__ void kernel_b (float *a)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    a[idx] = blockIdx.x;
}

__global__ void kernel_c (float *a)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    a[idx] = threadIdx.x;
}

int main(void)
{
    float *a_h, *b_h; // pointers to host memory
    float *a_d; // pointer to device memory
    float *k_a_h, *k_b_h, *k_c_h; // pointers to minimal kernel on host
    float *k_a_d, *k_b_d, *k_c_d; // pointers to minimal kernel on device
    int i, N = 15;
    size_t size = N*sizeof(float);
    // allocate arrays on host
    a_h = (float *)malloc(size);
    b_h = (float *)malloc(size);
    k_a_h = (float *)malloc(size);
    k_b_h = (float *)malloc(size);
    k_c_h = (float *)malloc(size);
    // allocate array on device
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &k_a_d, size);
    hipMalloc((void **) &k_b_d, size);
    hipMalloc((void **) &k_c_d, size);
    // initialization of host data
    for (i=0; i<N; i++) a_h[i] = (float)i;
    // copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(k_a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(k_b_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(k_c_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    // do calculation on host
    printf("HOST\n");
    printf("increment array on host\n");
    incrementArrayOnHost(a_h, N);
    // do calculation on device:
    printf("DEVICE\n");
    // Part 1 of 2. Compute execution configuration
    printf("compute execution configuration\n");
    int blockSize = 4;
    int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
    // Part 2 of 2. Call incrementArrayOnDevice kernel
    printf("increment array on device\n");
    incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
    // Retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

    printf("calculate modified kernel\n");
    int blockSize_mk = 5;
    int nBlocks_mk = N/blockSize_mk + (N%blockSize_mk == 0?0:1);
    kernel_a <<< nBlocks_mk, blockSize_mk >>> (k_a_d);
    kernel_b <<< nBlocks_mk, blockSize_mk >>> (k_b_d);
    kernel_c <<< nBlocks_mk, blockSize_mk >>> (k_c_d);
    hipMemcpy(k_a_h, k_a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(k_b_h, k_b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(k_c_h, k_c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

    // check results
    printf("HOSTS\tDEVICE\tKERNEL_a\tKERNEL_b\tKERNEL_c\n");
    for (i=0; i<N; i++)
    {
        printf("%.0f\t%.0f\t%.0f\t\t%.0f\t\t%.0f\n", a_h[i], b_h[i], k_a_h[i], k_b_h[i], k_c_h[i]);
        // assert(a_h[i] == b_h[i]);
    }
    // printf("passing assert so its valid!!!\n");
    // cleanup
    free(a_h); free(b_h); hipFree(a_d);
}