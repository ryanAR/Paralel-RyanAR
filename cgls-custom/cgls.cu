#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <cmath>

#include "cgls.cuh"

// Define real type.
typedef double real_t;
typedef hipDoubleComplex complex_t;
#define csr2csc hipsparseDcsr2csc
#define makeComplex make_hipDoubleComplex
// #define csr2csc hipsparseScsr2csc
// typedef float real_t;
// typedef hipFloatComplex complex_t;
// #define makeComplex make_hipFloatComplex

// Generates random CSR matrix with entries in [-1, 1]. The matrix will have
// exactly nnz non-zeros. All arrays must be pre-allocated.
void CsrMatGen(int m, int n, int nnz, real_t *val, int *rptr, int *cind) {
  real_t kRandMax = static_cast<real_t>(RAND_MAX);
  real_t kM = static_cast<real_t>(m);
  real_t kN = static_cast<real_t>(n);

  int num = 0;
  for (int i = 0; i < m; ++i) {
    rptr[i] = num;
    for (int j = 0; j < n && num < nnz; ++j) {
      if (rand() / kRandMax * ((kM - i) * kN - j) < (nnz - num)) {
        val[num] = 2 * (rand() - kRandMax / 2) / kRandMax;
        cind[num] = j;
        num++;
      }
    }
  }
  rptr[m] = nnz;
}

// Test CGLS on larger random matrix.
void test() {
  // Reset random seed.
  srand(0);

  // Initialize variables.
  real_t shift = 1;
  real_t tol = 1e-6;
  int maxit = 30;
  bool quiet = false;
  int m = 8096;
  int n = 8096;
  int nnz = 10000;

  printf("M = %d\n", m);
  printf("N = %d\n", n);

  // Initialize data.
  real_t *val_h = new real_t[nnz];
  int *cind_h = new int[nnz];
  int *rptr_h = new int[m + 1];
  real_t *b_h = new real_t[m];
  real_t *x1_h = new real_t[n]();
  real_t *x2_h = new real_t[n]();
  real_t *x3_h = new real_t[n]();
  real_t *x4_h = new real_t[n]();

  // Generate data.
  CsrMatGen(m, n, nnz, val_h, rptr_h, cind_h);
  for (int i = 0; i < m; ++i)
    b_h[i] = rand() / static_cast<real_t>(RAND_MAX);

  // Allocate x and b
  real_t *b_d, *x1_d, *x2_d, *x3_d, *x4_d;
  hipMalloc(&x1_d, n * sizeof(real_t));
  hipMalloc(&x2_d, n * sizeof(real_t));
  hipMalloc(&x3_d, n * sizeof(real_t));
  hipMalloc(&x4_d, n * sizeof(real_t));
  hipMalloc(&b_d, m * sizeof(real_t));

  // Allocate A
  real_t *val_a_d;
  int *cind_a_d, *rptr_a_d;
  hipMalloc(&val_a_d, nnz * sizeof(real_t));
  hipMalloc(&cind_a_d, nnz * sizeof(int));
  hipMalloc(&rptr_a_d, (m + 1) * sizeof(int));

  // Allocate A^T
  real_t *val_at_d;
  int *cind_at_d, *rptr_at_d;
  hipMalloc(&val_at_d, nnz * sizeof(real_t));
  hipMalloc(&cind_at_d, nnz * sizeof(int));
  hipMalloc(&rptr_at_d, (n + 1) * sizeof(int));

  // Transfer all data to device.
  hipMemcpy(b_d, b_h, m * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x1_d, x1_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x2_d, x2_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x3_d, x3_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x4_d, x4_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(val_a_d, val_h, nnz * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(cind_a_d, cind_h, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(rptr_a_d, rptr_h, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

  // Make A^T copy.
  hipsparseHandle_t handle_s;
  hipsparseCreate(&handle_s);
  csr2csc(handle_s, m, n, nnz, val_a_d, rptr_a_d, cind_a_d, val_at_d,
      cind_at_d, rptr_at_d, HIPSPARSE_ACTION_NUMERIC,
      HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  hipsparseDestroy(handle_s);

  // Solve with only A.
  int flag1 = cgls::Solve<real_t, cgls::CSR>(val_a_d, rptr_a_d, cind_a_d,
      m, n, nnz, b_d, x1_d, shift, tol, maxit, quiet);
  int flag2 = cgls::Solve<real_t, cgls::CSC>(val_at_d, rptr_at_d, cind_at_d,
      m, n, nnz, b_d, x2_d, shift, tol, maxit, quiet);

  // Solve with A and A^T.
  int flag3 = cgls::Solve<real_t, cgls::CSR>(val_a_d, rptr_a_d, cind_a_d,
      val_at_d, rptr_at_d, cind_at_d, m, n, nnz, b_d, x3_d, shift, tol, maxit,
      quiet);
  int flag4 = cgls::Solve<real_t, cgls::CSC>(val_at_d, rptr_at_d, cind_at_d,
      val_a_d, rptr_a_d, cind_a_d, m, n, nnz, b_d, x4_d, shift, tol, maxit,
      quiet);

  // Retrieve solution.
  hipMemcpy(x1_h, x1_d, n * sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(x2_h, x2_d, n * sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(x3_h, x3_d, n * sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(x4_h, x4_d, n * sizeof(real_t), hipMemcpyDeviceToHost);

  // Compute error and print.
  real_t err1 = 0, err2 = 0, err3 = 0;
  for (int i = 0; i < n; ++i)
    err1 += (x1_h[i] - x2_h[i]) * (x1_h[i] - x2_h[i]);
  err1 = std::sqrt(err1);
  for (int i = 0; i < n; ++i)
    err2 += (x1_h[i] - x3_h[i]) * (x1_h[i] - x3_h[i]);
  err2 = std::sqrt(err2);
  for (int i = 0; i < n; ++i)
    err3 += (x1_h[i] - x4_h[i]) * (x1_h[i] - x4_h[i]);
  err3 = std::sqrt(err3);

  if (flag1 == 0 && flag2 == 0 && flag3 == 0 && flag4 == 0
      && err1 < tol && err2 < tol && err3 < tol) {
    printf("Test Passed: Flag = (%d, %d, %d, %d), Error = (%e, %e, %e)\n",
        flag1, flag2, flag3, flag4, err1, err2, err3);
  } else {
    printf("Test Failed: Flag = (%d, %d, %d, %d), Error = (%e, %e, %e)\n",
        flag1, flag2, flag3, flag4, err1, err2, err3);
  }

  // Free data.
  hipFree(b_d);
  hipFree(x1_d);
  hipFree(x2_d);
  hipFree(x3_d);
  hipFree(x4_d);

  hipFree(val_a_d);
  hipFree(cind_a_d);
  hipFree(rptr_a_d);

  hipFree(val_at_d);
  hipFree(cind_at_d);
  hipFree(rptr_at_d);

  delete [] val_h;
  delete [] rptr_h;
  delete [] cind_h;
  delete [] x1_h;
  delete [] x2_h;
  delete [] b_h;
}

// Run tests.
int main() {
  // Execution time
  clock_t t_start = clock();

  // Run test
  test();

  // Execution time
  clock_t t_end = clock();
  double t = (double)(t_end-t_start) / (CLOCKS_PER_SEC/1000);
  printf("Execution time (milliseconds) = %f\n", t);
}

